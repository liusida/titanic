#include "hip/hip_runtime.h"
#include <cmy_contact_plane.h>

CMyContactPlane::CMyContactPlane(const Vec & normal, double offset) :
    ContactPlane(normal, offset),
    _color1({.5,.4,.5}),
    _color2({.5,.5,.4})
{
}

void CMyContactPlane::setPlaneColor(glm::vec3 color1, glm::vec3 color2) {
    _color1 = color1;
    _color2 = color2;
}

void CMyContactPlane::generateBuffers() {
    Vec temp = (dot(_normal, Vec(0, 1, 0)) < 0.8) ? Vec(0, 1, 0) : Vec(1, 0, 0);

    Vec v1 = cross(_normal, temp); // two unit vectors along plane
    v1 = v1 / v1.norm();

    Vec v2 = cross(_normal, v1);
    v2 = v2 / v2.norm();

    const static GLfloat vertex_buffer_platform[118] = {
            -1, -1, -1,
            -1, -1,  1,
            -1,  1,  1,
            1,  1, -1,
            -1, -1, -1,
            -1,  1, -1,
            1, -1,  1,
            -1, -1, -1,
            1, -1, -1,
            1,  1, -1,
            1, -1, -1,
            -1, -1, -1,
            -1, -1, -1,
            -1,  1,  1,
            -1,  1, -1,
            1, -1,  1,
            -1, -1,  1,
            -1, -1, -1,
            -1,  1,  1,
            -1, -1,  1,
            1, -1,  1,
            1,  1,  1,
            1, -1, -1,
            1,  1, -1,
            1, -1, -1,
            1,  1,  1,
            1, -1,  1,
            1,  1,  1,
            1,  1, -1,
            -1,  1, -1,
            1,  1,  1,
            -1,  1, -1,
            -1,  1,  1,
            1,  1,  1,
            -1,  1,  1,
            1, -1,  1
    };

    GLfloat vertex_data[108];

    for (int i = 0; i < 36; i++) {
        Vec temp = Vec(vertex_buffer_platform[3 * i], vertex_buffer_platform[3 * i + 1], vertex_buffer_platform[3 * i + 2]);
        Vec vertex = 30 * dot(v1, temp) * v1 + 30 * dot(v2, temp) * v2 + _normal * (_offset + dot(_normal, temp) - 1.0);

        vertex_data[3 * i] = vertex[0];
        vertex_data[3 * i + 1] = vertex[1];
        vertex_data[3 * i + 2] = vertex[2];
    }

    glGenBuffers(1, &vertices); // create buffer for these vertices
    glBindBuffer(GL_ARRAY_BUFFER, vertices);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertex_data), vertex_data, GL_STATIC_DRAW);

    GLfloat g_color_buffer_data[108];

    for (int i = 0; i < 36; i++) {
        if (i%2) {
            g_color_buffer_data[3 * i] = _color1[0];
            g_color_buffer_data[3 * i + 1] = _color1[1];
            g_color_buffer_data[3 * i + 2] = _color1[2];
        } else {
            g_color_buffer_data[3 * i] = _color2[0];
            g_color_buffer_data[3 * i + 1] = _color2[1];
            g_color_buffer_data[3 * i + 2] = _color2[2];
        }
    }

    glGenBuffers(1, &colors);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glBufferData(GL_ARRAY_BUFFER, sizeof(g_color_buffer_data), g_color_buffer_data, GL_STATIC_DRAW);

    _initialized = true;
}
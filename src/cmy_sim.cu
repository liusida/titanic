#include "hip/hip_runtime.h"
#include <cmy_sim.h>
#include <cmy_contact_plane.h>

#ifdef GRAPHICS
#include <GLFW/glfw3.h>
#endif

CMySimulation::CMySimulation()
{
    sky_color = {1.0f, 1.0f, 1.0f, 0.0f};
}

void CMySimulation::setSkyColor(Color color ) {
    sky_color = color;
}

void CMySimulation::start() {
    if (ENDED) {
        throw std::runtime_error("The simulation has ended. Cannot call sim.start() after the end of the simulation.");
    }

    if (masses.size() == 0) {
        throw std::runtime_error("No masses have been added. Please add masses before starting the simulation.");
    }

    std::cout << "Starting simulation with " << masses.size() << " masses and " << springs.size() << " springs." << std::endl;

    RUNNING = true;
    STARTED = true;

    T = 0;

    if (this -> dt == 0.0) { // if dt hasn't been set by the user.
        dt = 0.01; // min delta

        for (Mass * m : masses) {
            if (m -> dt < dt)
                dt = m -> dt;
        }
    }

#ifdef GRAPHICS // SDL2 window needs to be created here for Mac OS
#ifdef SDL2
    createSDLWindow();
#endif
#endif

    updateCudaParameters();

    d_constraints.d_balls = thrust::raw_pointer_cast(&d_balls[0]);
    d_constraints.d_planes = thrust::raw_pointer_cast(&d_planes[0]);
    d_constraints.num_balls = d_balls.size();
    d_constraints.num_planes = d_planes.size();

    update_constraints = false;

//    hipDeviceSetLimit(hipLimitMallocHeapSize, 5 * (masses.size() * sizeof(CUDA_MASS) + springs.size() * sizeof(CUDA_SPRING)));
    toArray();

    d_mass = thrust::raw_pointer_cast(d_masses.data());
    d_spring = thrust::raw_pointer_cast(d_springs.data());

    gpu_thread = std::thread(&CMySimulation::_run, this);
}

void CMySimulation::_run() { // repeatedly start next
#ifdef GRAPHICS

#ifndef SDL2 // GLFW window needs to be created here for Windows
    createGLFWWindow();
#endif

#ifdef SDL2
    SDL_GL_MakeCurrent(window, context);
#endif
    GLuint VertexArrayID;
    glGenVertexArrays(1, &VertexArrayID);
    glBindVertexArray(VertexArrayID);

//    glEnable(GL_LIGHTING);
//    glEnable(GL_LIGHT0);

    // Create and compile our GLSL program from the shaders
    this -> programID = LoadShaders(); // ("shaders/StandardShading.vertexshader", "shaders/StandardShading.fragmentshader"); //
    // Get a handle for our "MVP" uniform

    this -> MVP = getProjection(camera, looks_at, up); // compute perspective projection matrix

    this -> MatrixID = glGetUniformLocation(programID, "MVP"); // doesn't seem to be necessary

    generateBuffers(); // generate buffers for all masses and springs

    for (Constraint * c : constraints) { // generate buffers for constraint objects
        c -> generateBuffers();
    }

#endif
    
    execute();

    GPU_DONE = true;
}

void my_framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

void CMySimulation::createGLFWWindow() { // Copy from sim.cu and change sky color
    // Initialise GLFW
    if( !glfwInit() ) // TODO throw errors here
    {
        fprintf( stderr, "Failed to initialize GLFW\n" );
        getchar();
        exit(1);
    }

    glfwWindowHint(GLFW_SAMPLES, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE); // To make MacOS happy; should not be needed
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE); //We don't want the old OpenGL
    glfwWindowHint(GLFW_RESIZABLE, GL_TRUE);

    glfwSwapInterval(1);

    // Open a window and create its OpenGL context
    window = glfwCreateWindow(1024, 768, "CUDA Physics Simulation", NULL, NULL);

    if (window == NULL) {
        fprintf(stderr,
                "Failed to open GLFW window. If you have an Intel GPU, they are not 3.3 compatible. Try the 2.1 version of the tutorials.\n");
        getchar();
        glfwTerminate();
        exit(1);
    }

    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, my_framebuffer_size_callback);

    glEnable(GL_DEPTH_TEST);
    //    // Accept fragment if it closer to the camera than the former one
    glDepthFunc(GL_LESS);

    // Initialize GLEW
    glewExperimental = true; // Needed for core profile
    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "Failed to initialize GLEW\n");
        getchar();
        glfwTerminate();
        exit(1);
    }


    // Ensure we can capture the escape key being pressed below
    glfwSetInputMode(window, GLFW_STICKY_KEYS, GL_TRUE);

    // Dark blue background
    glClearColor(sky_color.r, sky_color.g, sky_color.b, sky_color.a);
    
}

void CMySimulation::createPlane(const Vec & abc, double d ) { // creates half-space ax + by + cz < d
    if (ENDED) {
        throw std::runtime_error("The simulation has ended. New objects cannot be created.");
    }

    CMyContactPlane * new_plane = new CMyContactPlane(abc, d);
    constraints.push_back(new_plane);
    d_planes.push_back(CudaContactPlane(*new_plane));

    update_constraints = true;
}

Constraint* CMySimulation::lastConstraint() {
    return constraints.back();
}

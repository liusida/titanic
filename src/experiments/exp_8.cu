#include "hip/hip_runtime.h"
/*
1. Sprinkle masses along the floor randomly.

2. Apply external forces according to an arbitrary 2d function/field (like a magnetic force) that makes them move along the floor in different directions and speeds according to the flow of the function.

3. When two masses touch there is a probability based on their velocities that they stick together permanently with a spring.
*/
#include <iostream>
using namespace std;
#include <sim.h>

#define random_double_number ((double) rand()/RAND_MAX)

__device__ Vec sam_field_force(CUDA_MASS * m) {
    Vec f = -0.1*m->pos;
    f[2] = 0;
    return f;
}
__device__ Vec gravity_force(CUDA_MASS * m) {
    return m->m * Vec(0,0,-9.81);
}
__device__ funcptr f_sam_field = sam_field_force ;
__device__ funcptr f_gravity = gravity_force;

int main() {
    funcptr h_sam_field;
    funcptr h_gravity;

    if (hipSuccess != hipMemcpyFromSymbol(&h_sam_field, HIP_SYMBOL(f_sam_field), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");
    if (hipSuccess != hipMemcpyFromSymbol(&h_gravity, HIP_SYMBOL(f_gravity), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");

    Simulation sim;
#ifdef GRAPHICS
    sim.setViewport(Vec(0,0,20), Vec(0,0,0), Vec(0,1,0));
#endif
    sim.setGlobalAcceleration(Vec(0,0,0));
    sim.createPlane(Vec(0,0,1), 0);
    sim.createField(h_sam_field);
    sim.createField(h_gravity);

    Mass * m1 = sim.createMass(Vec(1,0,0));
    Mass * m2 = sim.createMass(Vec(0,1,0));
    for (unsigned i=0;i<20;i++)
        sim.createMass(Vec(random_double_number*10,random_double_number*10,0));
    m1->vel = Vec(3,0,0);
    m2->vel = Vec(0,1,0);
    //sim.collision.strength = 0.0f;
    sim.start();
    while(1) {
        std::this_thread::sleep_for(std::chrono::microseconds(1));
        sim.processCollision();
    }
}
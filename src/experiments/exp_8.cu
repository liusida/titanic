#include "hip/hip_runtime.h"
/*
1. Sprinkle masses along the floor randomly.

2. Apply external forces according to an arbitrary 2d function/field (like a magnetic force) that makes them move along the floor in different directions and speeds according to the flow of the function.

3. When two masses touch there is a probability based on their velocities that they stick together permanently with a spring.
*/
#include <iostream>
using namespace std;
#include <sim.h>

#define random_double_number ((double) rand()/RAND_MAX)

__device__ Vec sam_field_force(CUDA_MASS * m) {
    Vec f = -0.1*m->pos;
    f[2] = 0;
    return f;
}
__device__ Vec gravity_force(CUDA_MASS * m) {
    return m->m * Vec(0,0,-9.81);
}
__device__ funcptr f_sam_field = sam_field_force ;
__device__ funcptr f_gravity = gravity_force;

int main() {
    funcptr h_sam_field;
    funcptr h_gravity;

    if (hipSuccess != hipMemcpyFromSymbol(&h_sam_field, HIP_SYMBOL(f_sam_field), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");
    if (hipSuccess != hipMemcpyFromSymbol(&h_gravity, HIP_SYMBOL(f_gravity), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");

    Simulation sim;
#ifdef GRAPHICS
    sim.setViewport(Vec(0,0,20), Vec(0,0,0), Vec(0,1,0));
#endif
    sim.setGlobalAcceleration(Vec(0,0,0));
    sim.createPlane(Vec(0,0,1), 0);
    sim.createField(h_sam_field);
    sim.createField(h_gravity);
    Mass * m_collision_1 = sim.createMass(Vec(0.1,0,0));
    Mass * m_collision_2 = sim.createMass(Vec(-0.1,0,0));
    Spring * s_collision = sim.createSpring(m_collision_1, m_collision_2);
    s_collision->_rest = (0.2); 

    Mass * m1 = sim.createMass(Vec(0,1,0));
    Mass * m2 = sim.createMass(Vec(0,1,0));
    // for (unsigned i=0;i<2;i++)
    //     sim.createMass(Vec(random_double_number*10,random_double_number*10,0));
    m1->vel = Vec(3,0,0);
    sim.start();
    // while(1) {
    //     sim.pause(sim.time()+1.3);
    //     sim.getAll();
    //     cout<<"Sim Time: "<<sim.time()<<endl;
    //     sim.printPositions();
    //     sim.resume();
    // }
}
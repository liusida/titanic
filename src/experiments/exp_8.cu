#include "hip/hip_runtime.h"
/*
1. Sprinkle masses along the floor randomly.

2. Apply external forces according to an arbitrary 2d function/field (like a magnetic force) that makes them move along the floor in different directions and speeds according to the flow of the function.

3. When two masses touch there is a probability based on their velocities that they stick together permanently with a spring.
*/
#include <iostream>
using namespace std;
#include <sim.h>

#define random_double_number ((double) rand()/RAND_MAX)

__device__ Vec sam_field_force(CUDA_MASS * m) {
    Vec f = -0.01*m->pos;
    f[2] = 0;
    return f;
}
__device__ Vec gravity_force(CUDA_MASS * m) {
    return m->m * Vec(0,0,-9.81);
}
__device__ funcptr f_sam_field = sam_field_force ;
__device__ funcptr f_gravity = gravity_force;

int main() {
    funcptr h_sam_field;
    funcptr h_gravity;

    if (hipSuccess != hipMemcpyFromSymbol(&h_sam_field, HIP_SYMBOL(f_sam_field), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");
    if (hipSuccess != hipMemcpyFromSymbol(&h_gravity, HIP_SYMBOL(f_gravity), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");

    Simulation sim;
#ifdef GRAPHICS
    sim.setViewport(Vec(0,0,40), Vec(0,0,0), Vec(0,1,0));
#endif
    sim.setGlobalAcceleration(Vec(0,0,0));
    sim.createPlane(Vec(0,0,1), 0);
    sim.createField(h_sam_field);
    sim.createField(h_gravity);

    for (unsigned i=0;i<20;i++) {
        Mass *m = sim.createMass(Vec(random_double_number*10,random_double_number*10,0));
        m->vel = Vec(random_double_number*10, random_double_number*10, 0);
        m->damping = 0.9998;
    }

    sim.minimum_distance = 2;

    sim.start();
    while(1) {
        std::this_thread::sleep_for(std::chrono::microseconds(10));
        sim.processCollision();
    }
}
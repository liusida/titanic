#include "hip/hip_runtime.h"
/*
1. Sprinkle masses along the floor randomly.

2. Apply external forces according to an arbitrary 2d function/field (like a magnetic force) that makes them move along the floor in different directions and speeds according to the flow of the function.

3. When two masses touch there is a probability based on their velocities that they stick together permanently with a spring.
*/
#include <iostream>
using namespace std;
#include <sim.h>

#define random_double_number ((double) rand()/RAND_MAX)

__device__ Vec sam_field_force(CUDA_MASS * m) {
    Vec f = -0.001*m->pos;
    f[2] = 0;
    return f;
}
__device__ Vec gravity_force(CUDA_MASS * m) {
    return m->m * Vec(0,0,-9.81);
}
__device__ funcptr f_sam_field = sam_field_force ;
__device__ funcptr f_gravity = gravity_force;

int main() {
    funcptr h_sam_field;
    funcptr h_gravity;

    if (hipSuccess != hipMemcpyFromSymbol(&h_sam_field, HIP_SYMBOL(f_sam_field), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");
    if (hipSuccess != hipMemcpyFromSymbol(&h_gravity, HIP_SYMBOL(f_gravity), sizeof (funcptr)))
        printf ("FAILED to get SYMBOL\n");

    Simulation sim;
#ifdef GRAPHICS
    sim.setViewport(Vec(0,0,30), Vec(0,0,0), Vec(0,1,0));
#endif
    sim.setGlobalAcceleration(Vec(0,0,0));
    sim.createPlane(Vec(0,0,1), 0);
    sim.createField(h_sam_field);
    sim.createField(h_gravity);

    for (unsigned i=0;i<100;i++) {
        Mass *m = sim.createMass(Vec(random_double_number*50-25,random_double_number*50-25,0));
        // m->vel[1] = m->pos[0]*0.2 + random_double_number;
        // m->vel[0] = -m->pos[1]*0.2 - random_double_number;
        // m->vel[2] = 0;
        //m->damping = 0.9999;
    }

    sim.minimum_distance = 1;

    sim.start();
    while(1) {
        std::this_thread::sleep_for(std::chrono::milliseconds(1));
        sim.processCollision();
    }
}
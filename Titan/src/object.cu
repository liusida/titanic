#include "hip/hip_runtime.h"
//
// Created by Jacob Austin on 5/21/18.
// object.cu defines constraint objects like planes and balls that allow the users
// to enforce limitations on movements of objects within the scene.
// Generally, an object defines the applyForce method that determines whether to apply a force
// to a mass, for example a normal force pushing the mass out of a constaint object or
// a frictional force.

#define GLM_FORCE_PURE
#include "object.h"
#include <cmath>
#include "sim.h"

#ifdef GRAPHICS
const Vec RED(1.0, 0.2, 0.2);
const Vec GREEN(0.2, 1.0, 0.2);
const Vec BLUE(0.2, 0.2, 1.0);
const Vec PURPLE(0.5, 0.2, 0.5);
#endif

__device__ const double NORMAL = 20000; // normal force coefficient for contact constaints
__device__ const double FRICTION_S = 1.0;  // static friction coeff rubber-on-concrete
__device__ const double FRICTION_K = 0.8;  // kinetic friction coeff

#ifdef CONSTRAINTS
void Container::addConstraint(CONSTRAINT_TYPE type, const Vec & v, double d) {
    for (Mass * m : masses) {
        m -> addConstraint(type, v, d);
    }
}

void Container::clearConstraints() {
    for (Mass * m : masses) {
        m -> clearConstraints();
    }
}

#endif

CUDA_CALLABLE_MEMBER CudaBall::CudaBall(const Vec & center, double radius) {
    _center = center;
    _radius = radius;
}

CUDA_CALLABLE_MEMBER CudaBall::CudaBall(const Ball & b) {
    _center = b._center;
    _radius = b._radius;
}

CUDA_CALLABLE_MEMBER void CudaBall::applyForce(CUDA_MASS * m) {
    double dist = (m -> pos - _center).norm();
    m -> force += (dist <= _radius) ? NORMAL * (m -> pos - _center) / dist : Vec(0, 0, 0);
}

CUDA_CALLABLE_MEMBER CudaContactPlane::CudaContactPlane(const Vec & normal, double offset) {
    _normal = normal / normal.norm();
    _offset = offset;
    _FRICTION_S = 0.0;
    _FRICTION_K = 0.0;
}

CudaContactPlane::CudaContactPlane(const ContactPlane & p) {
    _normal = p._normal;
    _offset = p._offset;

    _FRICTION_S = p._FRICTION_S;
    _FRICTION_K = p._FRICTION_K;
}

CUDA_CALLABLE_MEMBER void CudaContactPlane::applyForce(CUDA_MASS * m) {
    //    m -> force += (disp < 0) ? - disp * NORMAL * _normal : 0 * _normal; // TODO fix this for the host

    double disp = dot(m -> pos, _normal) - _offset; // displacement into the plane
    Vec f_normal = dot(m -> force, _normal) * _normal; // normal force

    if (disp < 0 && (_FRICTION_S > 0 || _FRICTION_K > 0)) { // if inside the plane
        Vec v_perp = m -> vel - dot(m -> vel, _normal) * _normal; // perpendicular velocity
        double v_norm = v_perp.norm();

        if (v_norm > 1e-16) { // kinetic friction domain
            double friction_mag = _FRICTION_K * f_normal.norm();
            m->force -= v_perp * friction_mag / v_norm;
        } else { // static friction
            Vec f_perp = m -> force - f_normal; // perpendicular force
	        if (_FRICTION_S * f_normal.norm() > f_perp.norm()) {
                m -> force -= f_perp;
	        } // else { // kinetic domain again
            //     double friction_mag = _FRICTION_K * f_normal.norm();
            //     m->force -= v_perp * friction_mag / v_norm;
	        // }
        }
    }

    // now apply the offset force to push the object out of the plane.
    Vec contact = (disp < 0) ? - disp * NORMAL * _normal : 0 * _normal; // displacement force
    double f_norm = contact.norm();
    m -> force += contact;

}

CUDA_CALLABLE_MEMBER CudaConstraintPlane::CudaConstraintPlane(const Vec & normal, double friction) {
    assert(normal.norm() != 0.0);

    _normal = normal / normal.norm();
    _friction = friction;
}

CUDA_CALLABLE_MEMBER void CudaConstraintPlane::applyForce(CUDA_MASS * m) {
    double normal_force = dot(m -> force, _normal);
    m -> force += - _normal * normal_force; // constraint force
    double v_norm = m -> vel.norm();

    if (v_norm >= 1e-16) {
        m -> vel += - _normal * dot(m -> vel, _normal); // constraint velocity
        m -> force += - _friction * normal_force * m -> vel / v_norm; // apply friction force
    }
}

CUDA_CALLABLE_MEMBER CudaDirection::CudaDirection(const Vec & tangent, double friction) {
    assert(tangent.norm() != 0.0);

    _tangent = tangent / tangent.norm();
    _friction = friction;
}

CUDA_CALLABLE_MEMBER void CudaDirection::applyForce(CUDA_MASS * m) {
    Vec normal_force = m -> force - dot(m -> force, _tangent) * _tangent;
    m -> force += - normal_force;

    if (m -> vel.norm() >= 1e-16) {
        m -> vel = _tangent * dot(m -> vel, _tangent);
        m -> force += - normal_force.norm() * _friction * _tangent;
    }
}

void Container::setMassValues(double m) { // set masses for all Mass objects
    for (Mass * mass : masses) {
        mass -> m += m;
    }
}

void Container::setSpringConstants(double k) {
    for (Spring * spring : springs) {
        spring -> _k = k;
    }
}

void Container::setDeltaT(double dt) { // set masses for all Mass objects
    for (Mass * mass : masses) {
        mass -> dt += dt;
    }
}

void Container::setRestLengths(double len) { // set masses for all Mass objects
    for (Spring * spring : springs) {
        spring -> _rest = len;
    }
}

void Container::add(Mass * m) {
    masses.push_back(m);
}

void Container::add(Spring * s) {
    springs.push_back(s);
}

void Container::add(Container * c) {
    for (Mass * m : c -> masses) {
        masses.push_back(m);
    }

    for (Spring * s : c -> springs) {
        springs.push_back(s);
    }
}

Cube::Cube(const Vec & center, double side_length) {
    _center = center;
    _side_length = side_length;

    for (int i = 0; i < 8; i++) {
        masses.push_back(new Mass(side_length * (Vec(i & 1, (i >> 1) & 1, (i >> 2) & 1) - Vec(0.5, 0.5, 0.5)) + center));
    }

    for (int i = 0; i < 8; i++) { // add the appropriate springs
        for (int j = i + 1; j < 8; j++) {
            springs.push_back(new Spring(masses[i], masses[j]));
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}

void Container::translate(const Vec & displ) {
    for (Mass * m : masses) {
        m -> pos += displ;
    }
}

void Container::rotate(const Vec & axis, double angle) {
    Vec com(0, 0, 0);

    double total_mass = 0;

    for (Mass * m : masses) {
        com += m -> m * m -> pos;
        total_mass += m -> m;
    }

    com = com / total_mass; // center of mass as centroid
    Vec temp_axis = axis / axis.norm();

    for (Mass * m : masses) {
        Vec temp = m -> pos - com; // subtract off center of mass
        Vec y = temp - dot(temp, temp_axis) * temp_axis; // project onto the given axis and find offset (y coordinate)

        if (y.norm() < 0.0001) { // if on the axis, don't do anything
            continue;
        }

        Vec planar(-sin(angle) * y.norm(), cos(angle) * y.norm(), 0); // coordinate in xy space
        Vec spatial = planar[0] * cross(temp_axis, y / y.norm()) + y / y.norm() * planar[1] + dot(temp, temp_axis) * temp_axis + com; // return to 3D space, then to COM space, then to absolute space

        m -> pos = spatial; // update position
    }
}

Lattice::Lattice(const Vec & center, const Vec & dims, int nx, int ny, int nz) {
    _center = center;
    _dims = dims;
    this -> nx = nx;
    this -> ny = ny;
    this -> nz = nz;

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                masses.push_back(new Mass(Vec((nx > 1) ? (double) i / (nx - 1.0) - 0.5 : 0, (ny > 1) ? j / (ny - 1.0) - 0.5 : 0, (nz > 1) ? k / (nz - 1.0) - 0.5 : 0) * dims + center));
            }
        }
    }

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                for (int l = 0; l < ((i != nx - 1) ? 2 : 1); l++) {
                    for (int m = 0; m < ((j != ny - 1) ? 2 : 1); m++) {
                        for (int n = 0; n < ((k != nz - 1) ? 2 : 1); n++) {
                            if (l != 0 || m != 0 || n != 0) {
                                springs.push_back(new Spring(masses[k + j * nz + i * ny * nz],
                                                             masses[(k + n) + (j + m) * nz + (i + l) * ny * nz]));
                            }
                        }
                    }
                }

                if (k != nz - 1) {
                    if (j != ny - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz], // get the full triangle
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                    }

                    if (i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }

                    if (j != ny - 1 && i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + (j + 1) * nz + (i + 1) * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + (i + 1) * ny * nz],
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + (j + 1) * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }
                }

                if (j != ny - 1 && i != nx - 1) {
                    springs.push_back(new Spring(masses[k + (j + 1) * nz + i * ny * nz],
                                                 masses[k + j * nz + (i + 1) * ny * nz]));
                }
            }
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}

#ifdef CONSTRAINTS
Beam::Beam(const Vec & center, const Vec & dims, int nx, int ny, int nz) {
    _center = center;
    _dims = dims;
    this -> nx = nx;
    this -> ny = ny;
    this -> nz = nz;

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
	            masses.push_back(new Mass(Vec((nx > 1) ? (double) i / (nx - 1.0) - 0.5 : 0, (ny > 1) ? j / (ny - 1.0) - 0.5 : 0, (nz > 1) ? k / (nz - 1.0) - 0.5 : 0) * dims + center));
                if (i == 0) {
                    masses[masses.size() - 1] -> constraints.fixed = true;
                }
            }
        }
    }

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                for (int l = 0; l < ((i != nx - 1) ? 2 : 1); l++) {
                    for (int m = 0; m < ((j != ny - 1) ? 2 : 1); m++) {
                        for (int n = 0; n < ((k != nz - 1) ? 2 : 1); n++) {
                            if (l != 0 || m != 0 || n != 0) {
                                springs.push_back(new Spring(masses[k + j * nz + i * ny * nz],
                                                             masses[(k + n) + (j + m) * nz + (i + l) * ny * nz]));
                            }
                        }
                    }
                }

                if (k != nz - 1) {
                    if (j != ny - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz], // get the full triangle
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                    }

                    if (i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }

                    if (j != ny - 1 && i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + (j + 1) * nz + (i + 1) * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + (i + 1) * ny * nz],
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + (j + 1) * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }
                }

                if (j != ny - 1 && i != nx - 1) {
                    springs.push_back(new Spring(masses[k + (j + 1) * nz + i * ny * nz],
                                                 masses[k + j * nz + (i + 1) * ny * nz]));
                }
            }
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}
#endif

Robot::Robot(const Vec & center, const cppn& encoding, double side_length,  double omega, double k_soft, double k_stiff){
    _center = center;
    _side_length = side_length;
    _omega = omega;
    _k_soft = k_soft;
    _k_stiff = k_stiff;
    _encoding = encoding;
    
    int RobotDim = encoding.size(); // number of cubes per side
    Vec dims(side_length,side_length,side_length);
    // keep trace of number of cubes that each mass is connected to 
    std::vector<std::vector<std::vector<int>>> mass_conn(RobotDim+1, std::vector<std::vector<int>>(RobotDim+1,std::vector<int>(RobotDim+1,0)));
    
    std::vector<std::vector<std::vector<Mass *>>> _masses(RobotDim+1, std::vector<std::vector<Mass *>>(RobotDim+1,std::vector<Mass *>(RobotDim+1,nullptr)));
  
    // store number of cubes that should be connected to each mass
    for (int i = 0; i < RobotDim+1; i++) {
      for (int j = 0; j < RobotDim+1; j++) {
	for (int k = 0; k < RobotDim+1; k++) {
	  // if index mode RobotDim+1 is 0, then it is on the edge
	  int i_edge = (i % (RobotDim)) ? 0:1; 
	  int j_edge = (j % (RobotDim)) ? 0:1;
	  int k_edge = (k % (RobotDim)) ? 0:1;

	
	  if (i_edge + j_edge + k_edge ==0){
	    mass_conn[i][j][k] = 8; //corner
	  }else if (i_edge+j_edge+k_edge ==3){
	    mass_conn[i][j][k] = 1; //corner
	  }else if (i_edge+j_edge+k_edge ==2){
	    mass_conn[i][j][k] = 2; //edge
	  }else{
	    mass_conn[i][j][k] = 4; //surface
	  }	
	}
      }
    }

    // Remove appropriate masses
    for (int i = 0; i < RobotDim; i++) {
      for (int j = 0; j < RobotDim; j++) {
	for (int k = 0; k < RobotDim; k++) {
	
	  int exist = encoding[i][j][k][0];

	  if (!exist){
	    // subtract connectedness of each mass for the cube
	    mass_conn[i][j][k] -= 1;
	    mass_conn[i][j][k+1] -= 1;
	    mass_conn[i][j+1][k] -= 1;
	    mass_conn[i][j+1][k+1] -= 1;
	    mass_conn[i+1][j][k] -= 1;
	    mass_conn[i+1][j][k+1] -= 1;
	    mass_conn[i+1][j+1][k] -= 1;
	    mass_conn[i+1][j+1][k+1] -= 1;
	  }
	}
      }
    }
  
    // create masses
    for (int i = 0; i < RobotDim+1; i++) {
        for (int j = 0; j < RobotDim+1; j++) {
            for (int k = 0; k < RobotDim + 1; k++) {
                if (mass_conn[i][j][k] > 0){
                    Mass * m;
                    if (RobotDim == 1) {
                    m = new Mass(Vec(i-0.5, j-0.5, k-0.5) * dims + _center);
                    } else {
                        m = new Mass(Vec(i / (RobotDim - 1.0) - 0.5,
                                j / (RobotDim - 1.0) - 0.5,
                                k / (RobotDim - 1.0) - 0.5) * dims + _center);
                    }

#ifdef GRAPHICS
                    m -> color = Vec(0,0,0);
#endif

                    masses.push_back(m);
                    _masses[i][j][k] = m;
                }
            }
        }
    }


    // create springs
    for (int i = 0; i < RobotDim; i++) {
        for (int j = 0; j < RobotDim; j++) {
	        for (int k = 0; k < RobotDim; k++) {
	
            int exist = encoding[i][j][k][0];

            if (exist) {
                int type = encoding[i][j][k][1];
            
                for(int l=0; l<8; l++) {
                    int l_x = (l<4)? 0:1;
                    int l_y = (l<2)? 0:(l<4)?1:(l<6)?0:1;
                    int l_z = (l%2)? 1:0;
                
                    for (int m=l+1; m<8; m++) {
                        int r_x = (m<4)? 0:1;
                        int r_y = (m<2)? 0:(m<4)?1:(m<6)?0:1;
                        int r_z = (m%2)? 1:0;

                        Spring * spr = new Spring(_masses[i+l_x][j+l_y][k+l_z],
                                    _masses[i+r_x][j+r_y][k+r_z]);
                        
                        spr -> _type = type;
                        spr -> _omega = omega;

                        if (type==0) { // green, contract then expand
                            spr -> _k = k_soft;

#ifdef GRAPHICS
                            _masses[i+l_x][j+l_y][k+l_z]->color += GREEN/16;
                            _masses[i+r_x][j+r_y][k+r_z]->color += GREEN/16;
#endif
                        } else if (type==1) { // red, expand then contract
                            spr -> _k = k_soft;
#ifdef GRAPHICS
                            _masses[i+l_x][j+l_y][k+l_z]->color += RED/16;
                            _masses[i+r_x][j+r_y][k+r_z]->color += RED/16;
#endif
                
                        } else if (type==2) { // passive soft
                            spr -> _k = k_soft;
#ifdef GRAPHICS
                            _masses[i+l_x][j+l_y][k+l_z]->color += BLUE/16;
                            _masses[i+r_x][j+r_y][k+r_z]->color += BLUE/16;
#endif
                        } else { // passive stiff
                            spr -> _k = k_stiff;
#ifdef GRAPHICS
                            _masses[i+l_x][j+l_y][k+l_z]->color += PURPLE/16;
                            _masses[i+r_x][j+r_y][k+r_z]->color += PURPLE/16;
#endif
                        }

		                springs.push_back(spr);
                        }
                    }
                }
            }
        }
    }

    
    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}

#ifdef CONSTRAINTS

void Container::makeFixed() {
    for (Mass * mass : masses) {
        mass -> constraints.fixed = true;
    }
}

LOCAL_CONSTRAINTS::LOCAL_CONSTRAINTS() {
//    constraint_plane = thrust::device_vector<CudaConstraintPlane>(1);
//    contact_plane = thrust::device_vector<CudaContactPlane>(1);
//    ball = thrust::device_vector<CudaBall>(1);
//    direction = thrust::device_vector<CudaDirection>(1);

//    contact_plane_ptr = thrust::raw_pointer_cast(contact_plane.data()); // TODO make sure this is safe
//    constraint_plane_ptr = thrust::raw_pointer_cast(constraint_plane.data());
//    ball_ptr = thrust::raw_pointer_cast(ball.data());
//    direction_ptr = thrust::raw_pointer_cast(direction.data());

    num_contact_planes = 0;
    num_constraint_planes = 0;
    num_balls = 0;
    num_directions = 0;

    drag_coefficient = 0;
    fixed = false;
}

CUDA_LOCAL_CONSTRAINTS::CUDA_LOCAL_CONSTRAINTS(LOCAL_CONSTRAINTS & c) {
    contact_plane = c.contact_plane_ptr;
    constraint_plane = c.constraint_plane_ptr;
    ball = c.ball_ptr;
    direction = c.direction_ptr;

    num_contact_planes = c.num_contact_planes;
    num_constraint_planes = c.num_constraint_planes;
    num_balls = c.num_balls;
    num_directions = c.num_directions;

    fixed = c.fixed;
    drag_coefficient = c.drag_coefficient;
}

#endif

#ifdef GRAPHICS

void Ball::normalize(GLfloat * v) {
    GLfloat norm = sqrt(pow(v[0], 2) + pow(v[1], 2) + pow(v[2],2)) / _radius;

    for (int i = 0; i < 3; i++) {
        v[i] /= norm;
    }
}

void Ball::writeTriangle(GLfloat * arr, GLfloat *v1, GLfloat *v2, GLfloat *v3) {
    for (int j = 0; j < 3; j++) {
        arr[j] = v1[j] + _center[j];
    }

    arr += 3;

    for (int j = 0; j < 3; j++) {
        arr[j] = v2[j] + _center[j];
    }

    arr += 3;

    for (int j = 0; j < 3; j++) {
        arr[j] = v3[j] + _center[j];
    }
}

void Ball::subdivide(GLfloat * arr, GLfloat *v1, GLfloat *v2, GLfloat *v3, int depth) {
    GLfloat v12[3], v23[3], v31[3];

    if (depth == 0) {
        writeTriangle(arr, v1, v2, v3);
        return;
    }

    for (int i = 0; i < 3; i++) {
        v12[i] = v1[i]+v2[i];
        v23[i] = v2[i]+v3[i];
        v31[i] = v3[i]+v1[i];
    }

    normalize(v12);
    normalize(v23);
    normalize(v31);

    subdivide(arr, v1, v12, v31, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v2, v23, v12, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v3, v31, v23, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v12, v23, v31, depth - 1);
}


void Ball::generateBuffers() {
    glm::vec3 color = {0.22f, 0.71f, 0.0f};

    GLfloat * vertex_data = new GLfloat[20 * 3 * 3 * (int) pow(4, depth)]; // times 4 for subdivision

    GLfloat X = (GLfloat) _radius * .525731112119133606;
    GLfloat Z = (GLfloat) _radius * .850650808352039932;

    static GLfloat vdata[12][3] = {
            {-X, 0.0, Z}, {X, 0.0, Z}, {-X, 0.0, -Z}, {X, 0.0, -Z},
            {0.0, Z, X}, {0.0, Z, -X}, {0.0, -Z, X}, {0.0, -Z, -X},
            {Z, X, 0.0}, {-Z, X, 0.0}, {Z, -X, 0.0}, {-Z, -X, 0.0}
    };
    static GLuint tindices[20][3] = {
            {0,4,1}, {0,9,4}, {9,5,4}, {4,5,8}, {4,8,1},
            {8,10,1}, {8,3,10}, {5,3,8}, {5,2,3}, {2,7,3},
            {7,10,3}, {7,6,10}, {7,11,6}, {11,0,6}, {0,1,6},
            {6,1,10}, {9,0,11}, {9,11,2}, {9,2,5}, {7,2,11} };

    for (int i = 0; i < 20; i++) {
        subdivide(&vertex_data[3 * 3 * (int) pow(4, depth) * i], vdata[tindices[i][0]], vdata[tindices[i][1]], vdata[tindices[i][2]], depth);
    }

    glGenBuffers(1, &vertices); // create buffer for these vertices
    glBindBuffer(GL_ARRAY_BUFFER, vertices);
    glBufferData(GL_ARRAY_BUFFER, 20 * 3 * 3 * (int) pow(4, depth) * sizeof(GLfloat), vertex_data, GL_STATIC_DRAW);

    GLfloat * color_data = new GLfloat[20 * 3 * 3 * (int) pow(4, depth)]; // TODO constant length array

    for (int i = 0; i < 20 * 3 * (int) pow(4, depth); i++) {
        color_data[3*i] = color[0];
        color_data[3*i + 1] = color[1];
        color_data[3*i + 2] = color[2];
    }

    glGenBuffers(1, &colors);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glBufferData(GL_ARRAY_BUFFER, 20 * 3 * 3 * (int) pow(4, depth) * sizeof(GLfloat), color_data, GL_STATIC_DRAW);

    delete [] color_data;
    delete [] vertex_data;

    _initialized = true;
}

void Ball::draw() {
    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vertices);

    glVertexAttribPointer(
            0,                  // attribute. No particular reason for 0, but must match the layout in the shader.
            3,                  // size
            GL_FLOAT,           // type
            GL_FALSE,           // normalized?
            0,                  // stride
            (void*)0            // array buffer offset
    );

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glVertexAttribPointer(
            1,                                // attribute. No particular reason for 1, but must match the layout in the shader.
            3,                                // size
            GL_FLOAT,                         // type
            GL_FALSE,                         // normalized?
            0,                                // stride
            (void*)0                          // array buffer offset
    );

    // Draw the triangle !
    glDrawArrays(GL_TRIANGLES, 0, 20 * 3 * (int) pow(4, depth)); // 12*3 indices starting at 0 -> 12 triangles

    glDisableVertexAttribArray(1);
    glDisableVertexAttribArray(0);
}

#endif

#ifdef GRAPHICS

void ContactPlane::generateBuffers() {
    glm::vec3 color = {0.22f, 0.71f, 0.0f};
    Vec temp = (dot(_normal, Vec(0, 1, 0)) < 0.8) ? Vec(0, 1, 0) : Vec(1, 0, 0);

    Vec v1 = cross(_normal, temp); // two unit vectors along plane
    v1 = v1 / v1.norm();

    Vec v2 = cross(_normal, v1);
    v2 = v2 / v2.norm();

    const static GLfloat vertex_buffer_platform[118] = {
            -1, -1, -1,
            -1, -1,  1,
            -1,  1,  1,
            1,  1, -1,
            -1, -1, -1,
            -1,  1, -1,
            1, -1,  1,
            -1, -1, -1,
            1, -1, -1,
            1,  1, -1,
            1, -1, -1,
            -1, -1, -1,
            -1, -1, -1,
            -1,  1,  1,
            -1,  1, -1,
            1, -1,  1,
            -1, -1,  1,
            -1, -1, -1,
            -1,  1,  1,
            -1, -1,  1,
            1, -1,  1,
            1,  1,  1,
            1, -1, -1,
            1,  1, -1,
            1, -1, -1,
            1,  1,  1,
            1, -1,  1,
            1,  1,  1,
            1,  1, -1,
            -1,  1, -1,
            1,  1,  1,
            -1,  1, -1,
            -1,  1,  1,
            1,  1,  1,
            -1,  1,  1,
            1, -1,  1
    };

    GLfloat vertex_data[108];

    for (int i = 0; i < 36; i++) {
        Vec temp = Vec(vertex_buffer_platform[3 * i], vertex_buffer_platform[3 * i + 1], vertex_buffer_platform[3 * i + 2]);
        Vec vertex = 10 * dot(v1, temp) * v1 + 10 * dot(v2, temp) * v2 + _normal * (_offset + dot(_normal, temp) - 1.0);

        vertex_data[3 * i] = vertex[0];
        vertex_data[3 * i + 1] = vertex[1];
        vertex_data[3 * i + 2] = vertex[2];
    }

    glGenBuffers(1, &vertices); // create buffer for these vertices
    glBindBuffer(GL_ARRAY_BUFFER, vertices);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertex_data), vertex_data, GL_STATIC_DRAW);

    GLfloat g_color_buffer_data[108];

    for (int i = 0; i < 36; i++) {
        g_color_buffer_data[3 * i] = color[0];
        g_color_buffer_data[3 * i + 1] = color[1];
        g_color_buffer_data[3 * i + 2] = color[2];
    }

    glGenBuffers(1, &colors);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glBufferData(GL_ARRAY_BUFFER, sizeof(g_color_buffer_data), g_color_buffer_data, GL_STATIC_DRAW);

    _initialized = true;
}

void ContactPlane::draw() {
    // 1st attribute buffer : vertices
    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vertices);

    glVertexAttribPointer(
            0,                  // attribute. No particular reason for 0, but must match the layout in the shader.
            3,                  // size
            GL_FLOAT,           // type
            GL_FALSE,           // normalized?
            0,                  // stride
            (void*)0            // array buffer offset
    );

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glVertexAttribPointer(
            1,                                // attribute. No particular reason for 1, but must match the layout in the shader.
            3,                                // size
            GL_FLOAT,                         // type
            GL_FALSE,                         // normalized?
            0,                                // stride
            (void*)0                          // array buffer offset
    );

    // Draw the triangle !
    glDrawArrays(GL_TRIANGLES, 0, 12*3); // 12*3 indices starting at 0 -> 12 triangles

    glDisableVertexAttribArray(1);
    glDisableVertexAttribArray(0);
}
#endif

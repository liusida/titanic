//
// Created by Jacob Austin on 5/17/18.
//
#define GLM_FORCE_PURE
#include "mass.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
    if (code != hipSuccess)
    {
        //fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort) {
            char buffer[200];
            snprintf(buffer, sizeof(buffer), "GPUassert error in CUDA kernel: %s %s %d\n", hipGetErrorString(code), file, line);
            std::string buffer_string = buffer;
            throw std::runtime_error(buffer_string);
            exit(code);
        }
    }
}

Mass::Mass() {
    m = 1.0;
    dt = 0.0001;
    damping = 1.0;
    T = 0;
    valid = true;
    arrayptr = nullptr;
    ref_count = 0;

#ifdef GRAPHICS
    color = Vec(1.0, 0.2, 0.2);
#endif
} // constructor TODO fix timing

void Mass::operator=(CUDA_MASS & mass) {
    m = mass.m;
    dt = mass.dt;
    T = mass.T;
    damping = mass.damping;
    pos = mass.pos;
    vel = mass.vel;
    acc = mass.acc;
    force = mass.force;
    valid = mass.valid;

    ref_count = this -> ref_count;
    arrayptr = this -> arrayptr;
    
    neighbors.clear();
    for (unsigned i=0;i<mass.num_neighbors;i++) {
        neighbors.push_back(mass.arrayptr->neighbors[i]);
    }
#ifdef CONSTRAINTS
    constraints = this -> constraints;
#endif

#ifdef GRAPHICS
    color = mass.color;
#endif
}

Mass::Mass(const Vec & position, double mass, bool fixed, double dt) {
    m = mass;
    pos = position;
    
    this -> dt = dt;

    T = 0;
    damping = 1.0;
    
    valid = true;
    arrayptr = nullptr;
    ref_count = 0;

#ifdef GRAPHICS
    color = Vec(1.0, 0.2, 0.2);
#endif
}

CUDA_MASS::CUDA_MASS(Mass &mass) {
    m = mass.m;
    dt = mass.dt;
    T = mass.T;
    damping = mass.damping;
    
    pos = mass.pos;
    vel = mass.vel;
    acc = mass.acc;
    force = mass.force;
    valid = true;

    arrayptr = &mass;
    //printf("CUDA: %p ->Mass: %p\n", this, arrayptr);
    CUDA_MASS ** temp = new CUDA_MASS * [mass.neighbors.size()];
    for (unsigned i=0;i<mass.neighbors.size();i++) {
        temp[i] = mass.neighbors[i]->arrayptr;
    }
    gpuErrchk(hipMalloc((void **) &neighbors, sizeof(CUDA_MASS *) * mass.neighbors.size()));
    gpuErrchk(hipMemcpy(neighbors, temp, mass.neighbors.size() * sizeof(CUDA_MASS *), hipMemcpyHostToDevice));
    delete temp;

    num_neighbors = mass.neighbors.size();
    //printf("CUDA_MASS(mass): # of neighbors: %d. \n", num_neighbors);

    // for (unsigned i=0;i<mass.neighbors.size();i++) {
    //     neighbors[i] = mass.neighbors[i]->arrayptr;
    //     printf("ptr: %p\n", mass.neighbors[i]->arrayptr);
    // }
#ifdef CONSTRAINTS
    constraints = CUDA_LOCAL_CONSTRAINTS(mass.constraints);
#endif

#ifdef GRAPHICS
    color = mass.color;
#endif
}

#ifdef CONSTRAINTS
void Mass::addConstraint(CONSTRAINT_TYPE type, const Vec & vec, double num) { // TODO make this more efficient
    if (type == 0) {
        this -> constraints.constraint_plane.push_back(CudaConstraintPlane(vec, num));
        this -> constraints.num_constraint_planes++;
        this -> constraints.constraint_plane_ptr = thrust::raw_pointer_cast(constraints.constraint_plane.data());
    } else if (type == 1) {
        this -> constraints.contact_plane.push_back(CudaContactPlane(vec, num));
        this -> constraints.num_contact_planes++;
        this -> constraints.contact_plane_ptr = thrust::raw_pointer_cast(constraints.contact_plane.data());
    } else if (type == 2) {
        this -> constraints.ball.push_back(CudaBall(vec, num));
        this -> constraints.num_balls++;
        this -> constraints.ball_ptr = thrust::raw_pointer_cast(constraints.ball.data());
    } else if (type == 3) {
        this -> constraints.direction.push_back(CudaDirection(vec, num));
        this -> constraints.num_directions++;
        this -> constraints.direction_ptr = thrust::raw_pointer_cast(constraints.direction.data());
    }
}

void Mass::clearConstraints(CONSTRAINT_TYPE type) {
    if (type == 0) {
        this -> constraints.constraint_plane.clear();
        this -> constraints.constraint_plane.shrink_to_fit();
        this -> constraints.num_constraint_planes = 0;
    } else if (type == 1) {
        this -> constraints.contact_plane.clear();
        this -> constraints.contact_plane.shrink_to_fit();
        this -> constraints.num_contact_planes = 0;
    } else if (type == 2) {
        this -> constraints.ball.clear();
        this -> constraints.ball.shrink_to_fit();
        this -> constraints.num_balls = 0;
    } else if (type == 3) {
        this -> constraints.direction.clear();
        this -> constraints.direction.shrink_to_fit();
        this -> constraints.num_directions = 0;
    }
}

void Mass::clearConstraints() {
    clearConstraints(CONSTRAINT_PLANE);
    clearConstraints(CONTACT_PLANE);
    clearConstraints(DIRECTION);
    clearConstraints(BALL);
}

void Mass::fix() {
    this -> constraints.fixed = true;
}
void Mass::unfix() {
    this -> constraints.fixed = false;
}

void Mass::setDrag(double C) {
    this -> constraints.drag_coefficient = C;
}


#endif

void Mass::decrementRefCount() {
    if (--ref_count == 0) {

        if (arrayptr) {
            hipFree(arrayptr);
        }

        delete this;
    }
}

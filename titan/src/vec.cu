#include "hip/hip_runtime.h"
//
//  vec.cpp
//  CUDA Physics
//
//  Created by Jacob Austin on 5/13/18.
//  Copyright © 2018 Jacob Austin. All rights reserved.
//

#include "vec.h"

#if __CUDA_ARCH__ < 600
__device__ double atomicDoubleAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

CUDA_DEVICE void Vec::atomicVecAdd(const Vec & v) {
atomicDoubleAdd(&data[0], (double) v.data[0]);
atomicDoubleAdd(&data[1], (double) v.data[1]);
atomicDoubleAdd(&data[2], (double) v.data[2]);
}

//Vec::Vec() {
//    data[0] = 0;
//    data[1] = 0;
//    data[2] = 0;
//}
//
//Vec::Vec(const Vec & v) {
//    data[0] = v.data[0];
//    data[1] = v.data[1];
//    data[2] = v.data[2];
//}
//
//Vec::Vec(double x, double y, double z) {
//    data[0] = x;
//    data[1] = y;
//    data[2] = z;
//}
//
//
//
//Vec & Vec::operator=(const Vec & v) {
//    if (this == &v) {
//        return *this;
//    }
//
//    data[0] = v.data[0];
//    data[1] = v.data[1];
//    data[2] = v.data[2];
//
//    return *this;
//}
//
//Vec & Vec::operator+=(const Vec & v) {
//    data[0] += v.data[0];
//    data[1] += v.data[1];
//    data[2] += v.data[2];
//    return *this;
//}
//
//Vec Vec::operator-() const{
//    return Vec(-data[0], -data[1], -data[2]);
//}
//
//
//Vec operator+(const Vec & v1, const Vec & v2) {
//    return Vec(v1.data[0] + v2.data[0], v1.data[1] + v2.data[1], v1.data[2] + v2.data[2]);
//}
//
//Vec operator-(const Vec & v1, const Vec & v2) {
//    return Vec(v1.data[0] - v2.data[0], v1.data[1] - v2.data[1], v1.data[2] - v2.data[2]);
//}
//
//Vec operator*(const double x, const Vec & v) {
//    return Vec(v.data[0] * x, v.data[1] * x, v.data[2] * x);
//}
//
//Vec operator*(const Vec & v, const double x) {
//    return x * v;
//}
//
//Vec operator*(const Vec & v1, const Vec & v2) {
//    return Vec(v1.data[0] * v2.data[0], v1.data[1] * v2.data[1], v1.data[2] * v2.data[2]);
//}
//
//Vec operator/(const Vec & v, const double x) {
//    return Vec(v.data[0] / x, v.data[1] / x, v.data[2] / x);
//}
//
////Vec operator/(const double x, const Vec & v) {
////    return x / v;
////}
//
//Vec operator/(const Vec & v1, const Vec & v2) {
//    return Vec(v1.data[0] / v2.data[0], v1.data[1] / v2.data[1], v1.data[2] / v2.data[2]);
//}
//
//std::ostream & operator << (std::ostream & strm, const Vec & v) {
//    return strm << "(" << v[0] << ", " << v[1] << ", " << v[2] << ")";
//}
//
//double & Vec::operator [] (int n) {
//    if (n < 0 || n >= 3) {
//        std::cerr << std::endl << "Out of bounds" << std::endl;
//        exit(EXIT_FAILURE);
//    } else {
//        return data[n];
//    }
//}
//
//const double & Vec::operator [] (int n) const {
//    if (n < 0 || n >= 3) {
//        std::cerr << std::endl << "Out of bounds" << std::endl;
//        exit(EXIT_FAILURE);
//    } else {
//        return data[n];
//    }
//}
//
//double Vec::norm() const {
//    return sqrt(pow(data[0], 2) + pow(data[1], 2) + pow(data[2], 2));
//}
//
//double Vec::sum() const {
//    return data[0] + data[1] + data[2];
//}

CUDA_CALLABLE_MEMBER double dot(const Vec & a, const Vec & b) {
    return (a * b).sum();
}

CUDA_CALLABLE_MEMBER Vec cross(const Vec &v1, const Vec &v2) {
    return Vec(v1[1] * v2[2] - v1[2] * v2[1], v2[0] * v1[2] - v1[0] * v2[2], v1[0] * v2[1] - v1[1] * v2[0]);
}